#include <iostream>

#include <hip/hip_runtime.h>

int _ConvertSMVer2Cores(int major, int minor) {
    // Refer to the CUDA Compute Capability documentation for the number of cores per multiprocessor
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
    switch ((major << 4) + minor) {
        case 0x10: return 8;  // Tesla
        case 0x11: return 8;  // Tesla
        case 0x12: return 8;  // Tesla
        case 0x13: return 8;  // Tesla
        case 0x20: return 32; // Fermi
        case 0x21: return 48; // Fermi
        case 0x30: return 192; // Kepler
        case 0x32: return 192; // Kepler
        case 0x35: return 192; // Kepler
        case 0x37: return 192; // Kepler
        case 0x50: return 128; // Maxwell
        case 0x52: return 128; // Maxwell
        case 0x53: return 128; // Maxwell
        case 0x60: return 64;  // Pascal
        case 0x61: return 128; // Pascal
        case 0x62: return 128; // Pascal
        case 0x70: return 64;  // Volta
        case 0x72: return 64;  // Volta
        case 0x75: return 64;  // Turing
        case 0x80: return 64;  // Ampere
        case 0x86: return 128;  // Ampere
        default: return -1;    // Unknown
    }
}


int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA-capable devices found." << std::endl;
        return 1;
    }

    for (int deviceId = 0; deviceId < deviceCount; ++deviceId) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, deviceId);

        std::cout << "Device #" << deviceId << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total Global Memory: " << deviceProp.totalGlobalMem << " bytes" << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  CUDA Cores per Multiprocessor: " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) << std::endl;
        std::cout << "  Total CUDA Cores: " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Clock Rate: " << deviceProp.clockRate << " kHz" << std::endl;
        std::cout << "  Memory Clock Rate: " << deviceProp.memoryClockRate << " kHz" << std::endl;
        std::cout << "  Memory Bus Width: " << deviceProp.memoryBusWidth << " bits" << std::endl;
        std::cout << "  L2 Cache Size: " << deviceProp.l2CacheSize << " bytes" << std::endl;
        std::cout << "  Warp Size: " << deviceProp.warpSize << " threads" << std::endl;
        std::cout << "  Maximum Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Maximum Block Dimensions: " << deviceProp.maxThreadsDim[0] << " x " << deviceProp.maxThreadsDim[1] << " x " << deviceProp.maxThreadsDim[2] << std::endl;
        std::cout << "  Maximum Grid Dimensions: " << deviceProp.maxGridSize[0] << " x " << deviceProp.maxGridSize[1] << " x " << deviceProp.maxGridSize[2] << std::endl;
        std::cout << "  Total Constant Memory: " << deviceProp.totalConstMem << " bytes" << std::endl;
        std::cout << "  Shared Memory per Block: " << deviceProp.sharedMemPerBlock << " bytes" << std::endl;
        std::cout << "  Max Registers per Block: " << deviceProp.regsPerBlock << std::endl;
        std::cout << "  Max Threads per Multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Concurrent Kernels: " << (deviceProp.concurrentKernels ? "Yes" : "No") << std::endl;
        std::cout << "  ECC Enabled: " << (deviceProp.ECCEnabled ? "Yes" : "No") << std::endl;
        std::cout << "  Async Engine Count: " << deviceProp.asyncEngineCount << std::endl;
        std::cout << "  Device Overlap: " << (deviceProp.deviceOverlap ? "Yes" : "No") << std::endl;
        std::cout << std::endl;
    }

    return 0;
}